
#include <hip/hip_runtime.h>
#include <stdio.h>

// 检查错误
#define cudaCheckErrors(msg)                                   \
    do                                                         \
    {                                                          \
        hipError_t __err = hipGetLastError();                \
        if (__err != hipSuccess)                              \
        {                                                      \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err),            \
                    __FILE__, __LINE__);                       \
            fprintf(stderr, "*** FAILED - ABORTING\n");        \
            exit(1);                                           \
        }                                                      \
    } while (0)

const int DSIZE = 32 * 1048576;
// 向量相加
__global__ void vadd(const float *A, const float *B, float *C, int ds)
{

    for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < ds; idx += gridDim.x * blockDim.x)
    {
        C[idx] = A[idx] + B[idx];
    }
}

int main()
{
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE];
    h_B = new float[DSIZE];
    h_C = new float[DSIZE];
    for (int i = 0; i < DSIZE; i++)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
        h_C[i] = 0;
    }
    hipMalloc(&d_A, DSIZE * sizeof(float));
    hipMalloc(&d_B, DSIZE * sizeof(float));
    hipMalloc(&d_C, DSIZE * sizeof(float));

    cudaCheckErrors("hipMalloc failure");

    hipMemcpy(d_A, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE * sizeof(float), hipMemcpyHostToDevice);

    cudaCheckErrors("cudamemcpy H2D failure");

    int blocks = 1;
    int threads = 1;
    vadd<<<blocks, threads>>>(d_A, d_B, d_C, DSIZE);

    cudaCheckErrors("kernel launch failure");
    hipMemcpy(h_C, d_C, DSIZE * sizeof(float), hipMemcpyDeviceToHost);

    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
    printf("A[0] = %f\n", h_A[0]);
    printf("B[0] = %f\n", h_B[0]);
    printf("C[0] = %f\n", h_C[0]);
    return 0;
}
