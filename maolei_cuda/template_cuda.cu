
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

// 将函数指针传递给 CUDA 内核的关键是使用指向设备指针的静态指针，
// 然后将指针复制到主机端。否则，我相信你会收到各种奇怪的错误。

// 定义函数指针类型
template<typename T>
using func_t = T (*)(T, T);

// 定义设备上的函数
template <typename T>
__device__ T add_func(T x,T y){
    return x + y;
}

template <typename T>
__device__ T mul_func(T x,T y){
    return x * y;
}

// 定义一个静态的设备指针
template <typename T> 
__device__ func_t<T> p_add_func = add_func<T>;
template <typename T> 
__device__ func_t<T> p_mul_func = mul_func<T>;

// 内核函数，接受函数指针
template <typename T> 
__global__ void kernel(func_t<T> op, T * d_x, T * d_y, T * result)
{
    *result = (*op)(*d_x, *d_y);
}

template <typename T> 
void test(T x, T y)
{
    func_t<T> h_add_func;
    func_t<T> h_mul_func;

    T * d_x, * d_y;
    hipMalloc(&d_x, sizeof(T));
    hipMalloc(&d_y, sizeof(T));
    hipMemcpy(d_x, &x, sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_y, &y, sizeof(T), hipMemcpyHostToDevice);

    T result;
    T * d_result, * h_result;
    hipMalloc(&d_result, sizeof(T));
    h_result = &result;

    // 将设备函数指针复制到主机端
    hipMemcpyFromSymbol(&h_add_func, HIP_SYMBOL(p_add_func<T>), sizeof(func_t<T>));
    hipMemcpyFromSymbol(&h_mul_func, HIP_SYMBOL(p_mul_func<T>), sizeof(func_t<T>));

    kernel<T><<<1,1>>>(h_add_func, d_x, d_y, d_result);
    hipDeviceSynchronize();
    hipMemcpy(h_result, d_result, sizeof(T), hipMemcpyDeviceToHost);
    std::cout << "Sum: " << result << std::endl;

    kernel<T><<<1,1>>>(h_mul_func, d_x, d_y, d_result);
    hipDeviceSynchronize();
    hipMemcpy(h_result, d_result, sizeof(T), hipMemcpyDeviceToHost);
    std::cout << "Product: " << result << std::endl;
}

int main()
{
    std::cout << "Test int for type int ..." << std::endl;
    test<int>(2.05, 10.00);

    std::cout << "Test float for type float ..." << std::endl;
    test<float>(2.05, 10.00);

    std::cout << "Test double for type double ..." << std::endl;
    test<double>(2.05, 10.00);
}